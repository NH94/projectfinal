

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>


using std::string;
using std::cout;
using std::endl;

#define INF 1000000
#define CHECK(call)                                                            \
              {                                                                              \
       const hipError_t error = call;                                            \
       if (error != hipSuccess)                                                  \
       {                                                                          \
              fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
              fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                            hipGetErrorString(error));                                    \
                            exit(1);                                                               \
       }                                                                          \
              }



namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
       std::cerr << msg << endl;
       abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
       return x * n + y;
}

int read_file(string filename) {
       std::ifstream inputf(filename, std::ifstream::in);
       if (!inputf.good()) {
              abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
       }
       inputf >> N;
       //input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
       assert(N < (1024 * 1024 * 20));
       mat = (int *) malloc(N * N * sizeof(int));
       for (int i = 0; i < N; i++)
              for (int j = 0; j < N; j++) {
                     inputf >> mat[convert_dimension_2D_1D(i, j, N)];
              }
       return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
       std::ofstream outputf("output.txt", std::ofstream::out);
       if (!has_negative_cycle) {
              for (int i = 0; i < N; i++) {
                     if (dist[i] > INF)
                            dist[i] = INF;
                     outputf << dist[i] << '\n';
              }
              outputf.flush();
       } else {
              outputf << "FOUND NEGATIVE CYCLE!" << endl;
       }
       outputf.close();
       return 0;
}
}//namespace utils


__global__ void bellman_ford_one_iter(int n, int *d_mat, int *d_dist, bool *d_has_next, int iter_num){
       int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
       int elementSkip = blockDim.x * gridDim.x;

       if(global_tid >= n) return;
       for(int u = 0 ; u < n ; u ++){
              for(int v = global_tid; v < n; v+= elementSkip){
                     int weight = d_mat[u * n + v];
                     if(weight < INF){
                            int new_dist = d_dist[u] + weight;
                            if(new_dist < d_dist[v]){
                                   d_dist[v] = new_dist;
                                   *d_has_next = true;
                            }
                     }
              }
       }

}
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
       dim3 blocks(blocksPerGrid);
       dim3 threads(threadsPerBlock);

       int iter_num = 0;
       int *d_mat, *d_dist;
       bool *d_has_next, h_has_next;

       hipMalloc(&d_mat, sizeof(int) * n * n);
       hipMalloc(&d_dist, sizeof(int) *n);
       hipMalloc(&d_has_next, sizeof(bool));


       *has_negative_cycle = false;

       for(int i = 0 ; i < n; i ++){
              dist[i] = INF;
       }

       dist[0] = 0;
       hipMemcpy(d_mat, mat, sizeof(int) * n * n, hipMemcpyHostToDevice);
       hipMemcpy(d_dist, dist, sizeof(int) * n, hipMemcpyHostToDevice);

       for(;;){
              h_has_next = false;
              hipMemcpy(d_has_next, &h_has_next, sizeof(bool), hipMemcpyHostToDevice);

              bellman_ford_one_iter<<<blocks, threads>>>(n, d_mat, d_dist, d_has_next, iter_num);
              CHECK(hipDeviceSynchronize());
              hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);

              iter_num++;
              if(iter_num >= n-1){
                     *has_negative_cycle = true;
                     break;
              }
              if(!h_has_next){
                     break;
              }

       }
       if(! *has_negative_cycle){
              hipMemcpy(dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);
       }

       hipFree(d_mat);
       hipFree(d_dist);
       hipFree(d_has_next);
}

int main(int argc, char **argv) {
       if (argc <= 1) {
              utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
       }
       if (argc <= 3) {
              utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
       }

       string filename = argv[1];
       int blockPerGrid = atoi(argv[2]);
       int threadsPerBlock = atoi(argv[3]);

       int *dist;
       bool has_negative_cycle = false;


       assert(utils::read_file(filename) == 0);
       dist = (int *) calloc(sizeof(int), utils::N);


       //time counter
       timeval start_wall_time_t, end_wall_time_t;
       float ms_wall;
       hipDeviceReset();
       //start timer
       gettimeofday(&start_wall_time_t, nullptr);
       //bellman-ford algorithm
       bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
       CHECK(hipDeviceSynchronize());
       //end timer
       gettimeofday(&end_wall_time_t, nullptr);
       ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
                     + end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

       std::cerr.setf(std::ios::fixed);
       std::cerr << std::setprecision(6) << "Time(s): " << (ms_wall/1000.0) << endl;
       utils::print_result(has_negative_cycle, dist);
       free(dist);
       free(utils::mat);

       return 0;
}
